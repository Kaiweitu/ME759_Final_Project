#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <cstdint>
#include <string>
#include "md5.cu"


// Convert Big endian to little endian
#define SWAP_INT32(x) (((x) >> 24) | (((x) & 0x00FF0000) >> 8) | (((x) & 0x0000FF00) << 8) | ((x) << 24))
#define CONST_MIN_PASSWORD_LENGTH 1
#define CONST_MAX_PASSWORD_LENGTH 7
#define CONST_CHAR_SET "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789"
#define CONST_CHAR_SET_LENGTH (sizeof(CONST_CHAR_SET) - 1)

using std::cout;
using std::cerr;
using std::endl;
using std::string;
using std::stoul;


__device__ __host__ bool getNextPw(char* pw_set, size_t step, size_t* length, size_t base = CONST_CHAR_SET_LENGTH) {
  size_t pos = 0;
  size_t carry = step;
  while (carry > 0 && pos < CONST_MAX_PASSWORD_LENGTH + 1) {
    size_t sum = carry + pw_set[pos];
    pw_set[pos] = sum % base;
    carry = sum / base;
    pos ++;
  }
  if (pos > *length)
    *length = pos;
  if (pos > CONST_MAX_PASSWORD_LENGTH)
    return false;
  return true;
}

__global__ void md5_attacker(size_t len, char* word_set, char *charset, char *cracked_pw, int hash_per_thread, uint32_t *target) {
  size_t idx = hash_per_thread * (blockIdx.x * blockDim.x + threadIdx.x) ;
  
  // Charset is shared by each block
  extern __shared__ char s[];
  uint32_t hash[4];
  uint32_t target_hash[4];
  char local_word_set[CONST_MAX_PASSWORD_LENGTH + 1];
  char local_word_text[CONST_MAX_PASSWORD_LENGTH + 1];
  
  // Copy from unified memory to local
  memcpy(target_hash, target, 4 * sizeof(uint32_t));
  memcpy(local_word_set, word_set, CONST_MAX_PASSWORD_LENGTH + 1);
  if (threadIdx.x == 0)
  memcpy(s, charset, sizeof(char) * CONST_CHAR_SET_LENGTH);
  
  // Synchronized here to ensure the shared variable is fully copied
  __syncthreads();
  if (!getNextPw(local_word_set, idx, &len)) return;
  for (size_t index = 0; index < hash_per_thread; index++) {
    for(size_t i = 0; i < len; i++){
      local_word_text[i] = s[local_word_set[i]];
    }
    
    // Calculate MD5 hashes
    md5((unsigned char*)local_word_text, len, hash);   
    bool isMatching = true;
    for (int j = 0; j < 4; j++) {
      if (hash[j] != target_hash[j]) {
        isMatching = false;
        break;
      }
    }
    // Find whether it's matching or nor
    if (isMatching) {
      memcpy(cracked_pw, local_word_text, len);
      return;
    } else if(!getNextPw(local_word_set, 1, &len)) 
      return;
  }
}

int main(int argc, char* argv[]) {
  if (argc != 5){
    cerr << "Error: Wrong number of argument" << endl;
    cout << "Usage: ./md5_attacker <md5_target> <block_num> <thread_num> <hash_per_thread>" << endl;
    return -1;
  } 

  string md5 = string(argv[1]);
  if (md5.length() != 32) {
    cerr << "Error: Incorrect length of target md5 hash value" << endl;
    return -1;
  }

  uint32_t *md5_target;
  char *pw_set, *cracked_pw, *char_set;

  // Allocated unified memory
  hipMallocManaged((void**)&md5_target, sizeof(uint32_t) * 4);
  hipMallocManaged((void**)&pw_set, sizeof(char) * (CONST_MAX_PASSWORD_LENGTH + 1));
  hipMallocManaged((void**)&cracked_pw, sizeof(char) * (CONST_MAX_PASSWORD_LENGTH + 1));
  hipMallocManaged((void**)&char_set, sizeof(char) * CONST_CHAR_SET_LENGTH);

  // Split the md5 hash into words and convert to little-endian
  for (size_t i = 0; i < 4; i++) {
    string hex_word = md5.substr(i * 8, 8);
    md5_target[i] = stoul(hex_word, 0, 16);
    // Convert from big-endian to little-endian 
    md5_target[i] = SWAP_INT32(md5_target[i]);
  }


  memset(pw_set, 0, CONST_MAX_PASSWORD_LENGTH + 1);
  memset(cracked_pw, 0, CONST_MAX_PASSWORD_LENGTH + 1);
  memcpy(char_set, CONST_CHAR_SET, CONST_CHAR_SET_LENGTH);
  
  size_t h_word_len = CONST_MIN_PASSWORD_LENGTH;

  const size_t block_num = stoul(argv[2]);
  const size_t thread_per_block = stoul(argv[3]);
  const size_t hash_per_thread = stoul(argv[4]);

  hipEvent_t start;
  hipEvent_t stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  while(true) {
    md5_attacker<<<block_num, thread_per_block, CONST_CHAR_SET_LENGTH>>>(h_word_len, pw_set, char_set, cracked_pw, hash_per_thread, md5_target);
    
    // wait to finish
    hipDeviceSynchronize();

    // Check whether this round has found the target
    if (*cracked_pw != 0) {     
      cout << cracked_pw << endl; 
      break;
    }

    // Update the pw set for next round
    if (!getNextPw(pw_set, thread_per_block * hash_per_thread * block_num, &h_word_len)) {
      cout << "Password Not Found" << endl;
      break;
    }
  }

  float ms;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  
  cout << ms << endl;
}
